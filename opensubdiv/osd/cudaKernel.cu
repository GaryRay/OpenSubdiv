#include "hip/hip_runtime.h"
//
//   Copyright 2013 Pixar
//
//   Licensed under the Apache License, Version 2.0 (the "Apache License")
//   with the following modification; you may not use this file except in
//   compliance with the Apache License and the following modification to it:
//   Section 6. Trademarks. is deleted and replaced with:
//
//   6. Trademarks. This License does not grant permission to use the trade
//      names, trademarks, service marks, or product names of the Licensor
//      and its affiliates, except as required to comply with Section 4(c) of
//      the License and to reproduce the content of the NOTICE file.
//
//   You may obtain a copy of the Apache License at
//
//       http://www.apache.org/licenses/LICENSE-2.0
//
//   Unless required by applicable law or agreed to in writing, software
//   distributed under the Apache License with the above modification is
//   distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
//   KIND, either express or implied. See the Apache License for the specific
//   language governing permissions and limitations under the Apache License.
//

#include <assert.h>

// -----------------------------------------------------------------------------
template<int N> struct DeviceVertex {

    float v[N];

    __device__ void addWithWeight(DeviceVertex<N> const & src, float weight) {
#pragma unroll
        for(int i = 0; i < N; ++i){
            v[i] += src.v[i] * weight;
        }
    }

    __device__ void clear() {
#pragma unroll
        for(int i = 0; i < N; ++i){
            v[i] = 0.0f;
        }
    }
};

// Specialize DeviceVertex for N=0 to avoid compile error:
// "flexible array member in otherwise empty struct"
template<> struct DeviceVertex<0> {
    __device__ void addWithWeight(DeviceVertex<0> &src, float weight) {}
    __device__ void clear() {}
};

// -----------------------------------------------------------------------------

__device__ void clear(float *dst, int count)
{
    for(int i = 0; i < count; ++i) dst[i] = 0;
}

__device__ void addWithWeight(float *dst, float const *src, float weight, int count)
{
    for(int i = 0; i < count; ++i) dst[i] += src[i] * weight;
}

// --------------------------------------------------------------------------------------------

template <int NUM_ELEMENTS> __global__ void
computeStencils(float const * cvs, float * vbuffer,
                unsigned char const * sizes,
                int const * offsets,
                int const * indices,
                float const * weights,
                int start, int end) {

    DeviceVertex<NUM_ELEMENTS> const * src =
        (DeviceVertex<NUM_ELEMENTS> const *)cvs;

    DeviceVertex<NUM_ELEMENTS> * verts =
        (DeviceVertex<NUM_ELEMENTS> *)vbuffer;

    int first = start + threadIdx.x + blockIdx.x*blockDim.x;

    for (int i=first; i<end; i += blockDim.x * gridDim.x) {

        int const * lindices = indices + offsets[i];
        float const * lweights = weights + offsets[i];

        DeviceVertex<NUM_ELEMENTS> dst;
        dst.clear();

        for (int j=0; j<sizes[i]; ++j) {
            dst.addWithWeight(src[lindices[j]], lweights[j]);
        }
        verts[i] = dst;
    }
}

__global__ void
computeStencils(float const * cvs, float * dst,
               int length, int stride,
               unsigned char const * sizes,
               int const * offsets,
               int const * indices,
               float const * weights,
               int start, int end) {

    int first = start + threadIdx.x + blockIdx.x*blockDim.x;

    for (int i=first; i<end; i += blockDim.x * gridDim.x) {

        int const * lindices = indices + offsets[i];
        float const * lweights = weights + offsets[i];

        float * dstVert = dst + i*stride;
        clear(dstVert, length);

        for (int j=0; j<sizes[i]; ++j) {

            float const * srcVert = cvs + lindices[j]*stride;

            addWithWeight(dstVert, srcVert, lweights[j], length);
        }
    }
}

// -----------------------------------------------------------------------------

#define USE_NVIDIA_OPTIMIZATION
#ifdef USE_NVIDIA_OPTIMIZATION

template< int NUM_ELEMENTS, int NUM_THREADS_PER_BLOCK >
__global__ void computeStencilsNv(float const *__restrict cvs,
                                  float * vbuffer,
                                  unsigned char const *__restrict sizes,
                                  int const *__restrict offsets,
                                  int const *__restrict indices,
                                  float const *__restrict weights,
                                  int start,
                                  int end)
{
  // Shared memory to stage indices/weights.
  __shared__ int   smem_indices_buffer[NUM_THREADS_PER_BLOCK];
  __shared__ float smem_weights_buffer[NUM_THREADS_PER_BLOCK];

  // The size of a single warp.
  const int WARP_SIZE = 32;
  // The number of warps per block.
  const int NUM_WARPS_PER_BLOCK = NUM_THREADS_PER_BLOCK / WARP_SIZE;
  // The number of outputs computed by a single warp.
  const int NUM_OUTPUTS_PER_WARP = WARP_SIZE / NUM_ELEMENTS;
  // The number of outputs computed by a block of threads.
  const int NUM_OUTPUTS_PER_BLOCK = NUM_OUTPUTS_PER_WARP*NUM_WARPS_PER_BLOCK;
  // The number of active threads in a warp.
  const int NUM_ACTIVE_THREADS_PER_WARP = NUM_OUTPUTS_PER_WARP * NUM_ELEMENTS;

  // The number of the warp inside the block.
  const int warpId = threadIdx.x / WARP_SIZE;
  const int laneId = threadIdx.x % WARP_SIZE;

  // We use NUM_ELEMENTS threads per output. Find which output/element a thread works on.
  int outputIdx = warpId*NUM_OUTPUTS_PER_WARP + laneId/NUM_ELEMENTS, elementIdx = laneId%NUM_ELEMENTS;

  // Each output corresponds to a section of shared memory.
  volatile int   *smem_indices = &smem_indices_buffer[warpId*WARP_SIZE + (laneId/NUM_ELEMENTS)*NUM_ELEMENTS];
  volatile float *smem_weights = &smem_weights_buffer[warpId*WARP_SIZE + (laneId/NUM_ELEMENTS)*NUM_ELEMENTS];

  // Disable threads that have nothing to do inside the warp.
  int i = end;
  if( laneId < NUM_ACTIVE_THREADS_PER_WARP )
    i = start + blockIdx.x*NUM_OUTPUTS_PER_BLOCK + outputIdx;

  // Iterate over the vertices.
  for( ; i < end ; i += gridDim.x*NUM_OUTPUTS_PER_BLOCK )
  {
    // Each thread computes an element of the final vertex.
    float x = 0.f;

    // Load the offset and the size for each vertex. We have NUM_THREADS_PER_VERTEX threads loading the same value.
    const int offset_i = offsets[i], size_i = sizes[i];

    // Iterate over the stencil.
    for( int j = offset_i, j_end = offset_i+size_i ; j < j_end ; )
    {
      int j_it = j + elementIdx;

      // Load some indices and some weights. The transaction is coalesced.
      smem_indices[elementIdx] = j_it < j_end ? indices[j_it] : 0;
      smem_weights[elementIdx] = j_it < j_end ? weights[j_it] : 0.f;

      // Thread now collaborates to load the vertices.
      #pragma unroll
      for( int k = 0 ; k < NUM_ELEMENTS ; ++k, ++j )
        if( j < j_end )
          x += smem_weights[k] * cvs[smem_indices[k]*NUM_ELEMENTS + elementIdx];
    }

    // Store the vertex.
    vbuffer[NUM_ELEMENTS*i + elementIdx] = x;
  }
}

template< int NUM_THREADS_PER_BLOCK >
__global__ void computeStencilsNv_v4(float const *__restrict cvs,
                                     float * vbuffer,
                                     unsigned char const *__restrict sizes,
                                     int const *__restrict offsets,
                                     int const *__restrict indices,
                                     float const *__restrict weights,
                                     int start,
                                     int end)
{
  // Iterate over the vertices.
  for( int i = start + blockIdx.x*NUM_THREADS_PER_BLOCK + threadIdx.x ; i < end ; i += gridDim.x*NUM_THREADS_PER_BLOCK )
  {
    // Each thread computes an element of the final vertex.
    float4 x = make_float4(0.f, 0.f, 0.f, 0.f);

    // Iterate over the stencil.
    for( int j = offsets[i], j_end = offsets[i]+sizes[i] ; j < j_end ; ++j )
    {
      float w = weights[j];
      float4 tmp = reinterpret_cast<const float4 *__restrict>(cvs)[indices[j]];
      x.x += w*tmp.x;
      x.y += w*tmp.y;
      x.z += w*tmp.z;
      x.w += w*tmp.w;
    }

    // Store the vertex.
    reinterpret_cast<float4*>(vbuffer)[i] = x;
  }
}

#endif // USE_NVIDIA_OPTIMIZATION

// -----------------------------------------------------------------------------

#include "../version.h"

#define OPT_KERNEL(NUM_ELEMENTS, KERNEL, X, Y, ARG) \
    if (length==NUM_ELEMENTS && stride==length) {   \
        KERNEL<NUM_ELEMENTS><<<X,Y>>>ARG;             \
        return;                                     \
    }

#ifdef USE_NVIDIA_OPTIMIZATION
#define OPT_KERNEL_NVIDIA(NUM_ELEMENTS, KERNEL, X, Y, ARG) \
    if (length==NUM_ELEMENTS && stride==length) {   \
        int gridDim = min(X, (end-start+Y-1)/Y); \
        KERNEL<NUM_ELEMENTS, Y><<<gridDim, Y>>>ARG; \
        return;                                     \
    }
#endif

extern "C" {

void
CudaComputeStencils(float const *cvs, float * dst,
                    int length, int stride,
                    unsigned char const * sizes,
                    int const * offsets,
                    int const * indices,
                    float const * weights,
                    int start, int end)
{
    assert(cvs and dst and sizes and offsets and indices and weights and (end>=start));

    if (length==0 or stride==0) {
        return;
    }

#ifdef USE_NVIDIA_OPTIMIZATION
    OPT_KERNEL_NVIDIA(3, computeStencilsNv, 2048, 256, (cvs, dst, sizes, offsets, indices, weights, start, end));
    //OPT_KERNEL_NVIDIA(4, computeStencilsNv, 2048, 256, (cvs, dst, sizes, offsets, indices, weights, start, end));
    if( length==4 && stride==length ) {
      int gridDim = min(2048, (end-start+256-1)/256);
      computeStencilsNv_v4<256><<<gridDim, 256>>>(cvs, dst, sizes, offsets, indices, weights, start, end);
      return;
    }
#else
    OPT_KERNEL(3, computeStencils, 512, 32, (cvs, dst, sizes, offsets, indices, weights, start, end));
    OPT_KERNEL(4, computeStencils, 512, 32, (cvs, dst, sizes, offsets, indices, weights, start, end));
#endif

    computeStencils <<<512, 32>>>(cvs, dst, length, stride,
        sizes, offsets, indices, weights, start, end);
}

// -----------------------------------------------------------------------------

}  /* extern "C" */
